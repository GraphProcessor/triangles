#include "hip/hip_runtime.h"
#include "gpu.h"

#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <iostream>
#include <vector>
#include <utility>
using namespace std;

__global__ void CalculatePointers(int n, int m, int* edges, int* pointers) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i < m; i += step) {
    int prev = i > 0 ? edges[2 * (i - 1)] : -1;
    int next = edges[2 * i];
    for (int j = prev + 1; j <= next; ++j)
      pointers[j] = i;
  }
  if (from == 0) pointers[n] = m;
}

void CudaAssert(hipError_t status, const char* code, const char* file, int l) {
  if (status == hipSuccess) return;
  cerr << "Cuda error: " << code << ", file " << file << ", line " << l << endl;
  exit(1);
}

#define CUCHECK(x) CudaAssert(x, #x, __FILE__, __LINE__)

uint64_t GpuEdgeIterator(const Edges& unordered_edges) {
  //Timer timer;
  
  const int n = NumVertices(unordered_edges);
  const int m = unordered_edges.size();

  //Log() << "Calc num vertices " << timer.SinceLast();
  
  //Log() << "Device initialization " << timer.SinceLast();

  int* dev_edges;
  int* dev_pointers;
  int* dev_results;

  CUCHECK(hipMalloc(&dev_edges, m * 2 * sizeof(int)));
  CUCHECK(hipMalloc(&dev_pointers, (n + 1) * sizeof(int)));
  CUCHECK(hipMalloc(&dev_results, m * sizeof(int)));

  //Log() << "hipMalloc " << timer.SinceLast();

  cerr << "Malloc done" << endl;

  CUCHECK(hipMemcpyAsync(dev_edges, unordered_edges.data(),
                          m * 2 * sizeof(int),
                          hipMemcpyHostToDevice));

  cerr << "Memcpy done" << endl;

  
  thrust::sort(thrust::device_ptr<uint64_t>((uint64_t*)dev_edges),
               thrust::device_ptr<uint64_t>((uint64_t*)dev_edges + m));
  

  cerr << "Sort done" << endl;

  CalculatePointers<<<(m + 256 - 1) / 256, 256>>>(n, m, dev_edges, dev_pointers);

  CUCHECK(hipDeviceSynchronize());

  cerr << "Calc ptrs kernel done" << endl;



  CUCHECK(hipFree(dev_edges));
  CUCHECK(hipFree(dev_pointers));
  CUCHECK(hipFree(dev_results));
  
  cerr << "Free done" << endl;

  return 0;
}

