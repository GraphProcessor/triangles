#include "hip/hip_runtime.h"
#include "gpu.h"

#include "gpu-thrust.h"
#include "timer.h"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <utility>
using namespace std;

#define NUM_BLOCKS 42
#define NUM_THREADS 128
#define WARP_SIZE 4

__global__ void CalculatePointers(int n, int m, int* edges, int* pointers) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i < m; i += step) {
    int prev = i > 0 ? edges[2 * (i - 1) + 1] : -1;
    int next = edges[2 * i + 1];
    for (int j = prev + 1; j <= next; ++j)
      pointers[j] = i;
  }
  if (from == 0) pointers[n] = m;
}

__global__ void CalculateFlags(int m, int* edges, int* pointers, int* flags) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i < m; i += step) {
    int a = edges[2 * i];
    int b = edges[2 * i + 1];
    int deg_a = pointers[a + 1] - pointers[a];
    int deg_b = pointers[b + 1] - pointers[b];
    flags[i] = (deg_a < deg_b) || (deg_a == deg_b && a < b);
  }
}

__global__ void UnzipEdges(int m, int* edges, int* unzipped_edges) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i < m; i += step) {
    unzipped_edges[i] = edges[2 * i];
    unzipped_edges[m + i] = edges[2 * i + 1];
  }
}

__global__ void CalculateTriangles(
    int m, int* edges, int* pointers, int* results) {
  int from = (NUM_THREADS * blockIdx.x + threadIdx.x) / WARP_SIZE;
  int step = NUM_BLOCKS * NUM_THREADS / WARP_SIZE;
  for (int i = from; i < m; i += step) {
    int u = edges[i], v = edges[m + i];

    int u_it = pointers[u], u_end = pointers[u + 1];
    int v_it = pointers[v], v_end = pointers[v + 1];
    int count = 0;

    while (u_it < u_end && v_it < v_end) {
      int a = edges[u_it], b = edges[v_it];
      if (a < b) {
        ++u_it;
      } else if (a > b) {
        ++v_it;
      } else {
        ++count;
        ++u_it;
        ++v_it;
      }
    }
    results[i] = count;
  }
}

void CudaAssert(hipError_t status, const char* code, const char* file, int l) {
  if (status == hipSuccess) return;
  cerr << "Cuda error: " << code << ", file " << file << ", line " << l << endl;
  exit(1);
}

#define CUCHECK(x) CudaAssert(x, #x, __FILE__, __LINE__)

uint64_t GpuEdgeIterator(const Edges& unordered_edges) {
  Timer* timer = Timer::NewTimer();
  
  int n = NumVertices(unordered_edges);
  int m = unordered_edges.size();

  Log() << "Calculate num vertices " << timer->SinceLast();

  int* dev_edges;
  int* dev_edges_unzipped;
  int* dev_pointers;
  int* dev_results;

  CUCHECK(hipMalloc(&dev_edges, m * 2 * sizeof(int)));
  CUCHECK(hipMalloc(&dev_edges_unzipped, m * 2 * sizeof(int)));
  CUCHECK(hipMalloc(&dev_pointers, (n + 1) * sizeof(int)));
  CUCHECK(hipMalloc(&dev_results, m * sizeof(int)));
  Log() << "Malloc " << timer->SinceLast();


  CUCHECK(hipMemcpyAsync(dev_edges, unordered_edges.data(),
                          m * 2 * sizeof(int),
                          hipMemcpyHostToDevice));
  CUCHECK(hipDeviceSynchronize());
  Log() << "Memcpy " << timer->SinceLast();

  SortEdges(m, dev_edges);
  CUCHECK(hipDeviceSynchronize());
  Log() << "Sort edges " << timer->SinceLast();

  CalculatePointers<<<NUM_BLOCKS, NUM_THREADS>>>(n, m, dev_edges, dev_pointers);
  CUCHECK(hipDeviceSynchronize());
  Log() << "Calculate ptrs 1 " << timer->SinceLast();

  CalculateFlags<<<NUM_BLOCKS, NUM_THREADS>>>(
      m, dev_edges, dev_pointers, dev_results);
  RemoveMarkedEdges(m, dev_edges, dev_results);
  CUCHECK(hipDeviceSynchronize());
  Log() << "Remove backward edges " << timer->SinceLast();

  m /= 2;
 
  CalculatePointers<<<NUM_BLOCKS, NUM_THREADS>>>(n, m, dev_edges, dev_pointers);
  CUCHECK(hipDeviceSynchronize());
  Log() << "Calculate ptrs 2 " << timer->SinceLast();

  UnzipEdges<<<NUM_BLOCKS, NUM_THREADS>>>(m, dev_edges, dev_edges_unzipped);
  CUCHECK(hipDeviceSynchronize());
  Log() << "Unzip edges " << timer->SinceLast();

  hipProfilerStart();
  CalculateTriangles<<<NUM_BLOCKS, NUM_THREADS>>>(
      m, dev_edges_unzipped, dev_pointers, dev_results);
  CUCHECK(hipDeviceSynchronize());
  hipProfilerStop();
  Log() << "Calculate triangles " << timer->SinceLast();

  uint64_t result = 0;
  result = SumResults(m, dev_results);
  Log() << "Reduce " << timer->SinceLast();

  CUCHECK(hipFree(dev_edges));
  CUCHECK(hipFree(dev_edges_unzipped));
  CUCHECK(hipFree(dev_pointers));
  CUCHECK(hipFree(dev_results));
  Log() << "Free " << timer->SinceLast();

  delete timer;

  return result;
}
