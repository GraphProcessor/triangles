#include "hip/hip_runtime.h"
#include "gpu.h"

#include "gpu-thrust.h"
#include "timer.h"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <utility>
using namespace std;

#define NUM_BLOCKS 42
#define NUM_THREADS 128
#define WARP_SIZE 4
#define NUM_WORKERS (NUM_BLOCKS * NUM_THREADS / WARP_SIZE)

__global__ void CalculatePointers(int n, int m, int* edges, int* pointers) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i < m; i += step) {
    int prev = i > 0 ? edges[2 * (i - 1) + 1] : -1;
    int next = edges[2 * i + 1];
    for (int j = prev + 1; j <= next; ++j)
      pointers[j] = i;
  }
  if (from == 0) pointers[n] = m;
}

__global__ void CalculateFlags(int m, int* edges, int* pointers, bool* flags) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i < m; i += step) {
    int a = edges[2 * i];
    int b = edges[2 * i + 1];
    int deg_a = pointers[a + 1] - pointers[a];
    int deg_b = pointers[b + 1] - pointers[b];
    flags[i] = (deg_a < deg_b) || (deg_a == deg_b && a < b);
  }
}

__global__ void UnzipEdges(int m, int* edges, int* unzipped_edges) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i < m; i += step) {
    unzipped_edges[i] = edges[2 * i];
    unzipped_edges[m + i] = edges[2 * i + 1];
  }
}

__global__ void CalculateTriangles(
    int m, int* edges, int* pointers, uint64_t* results) {
  int from = (NUM_THREADS * blockIdx.x + threadIdx.x) / WARP_SIZE;
  int step = NUM_WORKERS;
  uint64_t count = 0;
  for (int i = from; i < m; i += step) {
    int u = edges[i], v = edges[m + i];

    int u_it = pointers[u], u_end = pointers[u + 1];
    int v_it = pointers[v], v_end = pointers[v + 1];

    while (u_it < u_end && v_it < v_end) {
      int a = edges[u_it], b = edges[v_it];
      if (a < b) {
        ++u_it;
      } else if (a > b) {
        ++v_it;
      } else {
        ++u_it;
        ++v_it;
        ++count;
      }
    }
  }

  results[from] = count;
}

void CudaAssert(hipError_t status, const char* code, const char* file, int l) {
  if (status == hipSuccess) return;
  cerr << "Cuda error: " << code << ", file " << file << ", line " << l << endl;
  exit(1);
}

#define CUCHECK(x) CudaAssert(x, #x, __FILE__, __LINE__)

uint64_t GpuEdgeIterator(const Edges& unordered_edges) {
  Timer* timer = Timer::NewTimer();
  
  int n = NumVertices(unordered_edges);
  int m = unordered_edges.size();

  timer->Done("Calculate number of vertices");

  int* dev_edges;
  int* dev_edges_unzipped;
  bool* dev_flags;
  int* dev_pointers;
  uint64_t* dev_results;

  CUCHECK(hipMalloc(&dev_edges, m * 2 * sizeof(int)));
  CUCHECK(hipMalloc(&dev_edges_unzipped, m * 2 * sizeof(int)));
  CUCHECK(hipMalloc(&dev_flags, m * sizeof(bool)));
  CUCHECK(hipMalloc(&dev_pointers, (n + 1) * sizeof(int)));
  CUCHECK(hipMalloc(&dev_results, NUM_WORKERS * sizeof(uint64_t)));
  timer->Done("Malloc");


  CUCHECK(hipMemcpyAsync(dev_edges, unordered_edges.data(),
                          m * 2 * sizeof(int),
                          hipMemcpyHostToDevice));
  CUCHECK(hipDeviceSynchronize());
  timer->Done("Memcpy");

  SortEdges(m, dev_edges);
  CUCHECK(hipDeviceSynchronize());
  timer->Done("Sort edges");

  CalculatePointers<<<NUM_BLOCKS, NUM_THREADS>>>(n, m, dev_edges, dev_pointers);
  CUCHECK(hipDeviceSynchronize());
  timer->Done("Calculate pointers 1");

  CalculateFlags<<<NUM_BLOCKS, NUM_THREADS>>>(
      m, dev_edges, dev_pointers, dev_flags);
  RemoveMarkedEdges(m, dev_edges, dev_flags);
  CUCHECK(hipDeviceSynchronize());
  timer->Done("Remove backward edges");

  m /= 2;
 
  CalculatePointers<<<NUM_BLOCKS, NUM_THREADS>>>(n, m, dev_edges, dev_pointers);
  CUCHECK(hipDeviceSynchronize());
  timer->Done("Calculate pointers 2");

  UnzipEdges<<<NUM_BLOCKS, NUM_THREADS>>>(m, dev_edges, dev_edges_unzipped);
  CUCHECK(hipDeviceSynchronize());
  timer->Done("Unzip edges");

  hipProfilerStart();
  CalculateTriangles<<<NUM_BLOCKS, NUM_THREADS>>>(
      m, dev_edges_unzipped, dev_pointers, dev_results);
  CUCHECK(hipDeviceSynchronize());
  hipProfilerStop();
  timer->Done("Calculate triangles");

  uint64_t result = 0;
  result = SumResults(NUM_WORKERS, dev_results);
  timer->Done("Reduce");

  CUCHECK(hipFree(dev_edges));
  CUCHECK(hipFree(dev_edges_unzipped));
  CUCHECK(hipFree(dev_flags));
  CUCHECK(hipFree(dev_pointers));
  CUCHECK(hipFree(dev_results));
  timer->Done("Free");

  delete timer;

  return result;
}
