#include "hip/hip_runtime.h"
#include "gpu.h"

#include "timer.h"

#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>

#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <utility>
using namespace std;

struct IsBackward {
  __host__ __device__ bool operator()(const unsigned long long edge) {
    int a = edge >> 32;
    int b = (edge << 32) >> 32;
    return a < b;
  }
};

__global__ void CalculatePointers(int n, int m, int* edges, int* pointers) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i < m; i += step) {
    int prev = i > 0 ? edges[2 * (i - 1) + 1] : -1;
    int next = edges[2 * i + 1];
    for (int j = prev + 1; j <= next; ++j)
      pointers[j] = i;
  }
  if (from == 0) pointers[n] = m;
}

__global__ void CalculateTriangles(int m, int* edges, int* pointers, int* results) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i < m; i += step) {
    int u = edges[2 * i], v = edges[2 * i + 1];
    int u_start = pointers[u], u_end = pointers[u + 1];
    int v_start = pointers[v], v_end = pointers[v + 1];
    int u_it = u_start, v_it = v_start;
    int count = 0;
    while (u_it < u_end && v_it < v_end) {
      int a = edges[2 * u_it], b = edges[2 * v_it];
      if (a < b) {
        ++u_it;
      } else if (a > b) {
        ++v_it;
      } else {
        ++count;
        ++u_it;
        ++v_it;
      }
    }
    results[i] = count;
  }
}

void CudaAssert(hipError_t status, const char* code, const char* file, int l) {
  if (status == hipSuccess) return;
  cerr << "Cuda error: " << code << ", file " << file << ", line " << l << endl;
  exit(1);
}

#define CUCHECK(x) CudaAssert(x, #x, __FILE__, __LINE__)

uint64_t GpuEdgeIterator(const Edges& unordered_edges) {
  Timer* timer = Timer::NewTimer();
  
  int n = NumVertices(unordered_edges);
  int m = unordered_edges.size();

  Log() << "Calc num vertices " << timer->SinceLast();
  
  int* dev_edges;
  int* dev_pointers;
  int* dev_results;

  CUCHECK(hipMalloc(&dev_edges, m * 2 * sizeof(int)));
  CUCHECK(hipMalloc(&dev_pointers, (n + 1) * sizeof(int)));
  CUCHECK(hipMalloc(&dev_results, m * sizeof(int)));

  Log() << "hipMalloc " << timer->SinceLast();


  CUCHECK(hipMemcpyAsync(dev_edges, unordered_edges.data(),
                          m * 2 * sizeof(int),
                          hipMemcpyHostToDevice));

  CUCHECK(hipDeviceSynchronize());

  Log() << "Memcpy done " << timer->SinceLast();

  thrust::remove_if(
     thrust::device_ptr<uint64_t>((uint64_t*)dev_edges),
     thrust::device_ptr<uint64_t>((uint64_t*)dev_edges + m),
     IsBackward());

  m /= 2;

  thrust::sort(
    thrust::device_ptr<uint64_t>((uint64_t*)dev_edges),
    thrust::device_ptr<uint64_t>((uint64_t*)dev_edges + m));
  
  CUCHECK(hipDeviceSynchronize());

  Log() << "Sort done " << timer->SinceLast();

  CalculatePointers<<<48, 256>>>(n, m, dev_edges, dev_pointers);

  CUCHECK(hipDeviceSynchronize());

  Log() << "Calc ptrs kernel done " << timer->SinceLast();

  CalculateTriangles<<<48, 256>>>(m, dev_edges, dev_pointers, dev_results);

  CUCHECK(hipDeviceSynchronize());

  Log() << "Calc tri kernel done " << timer->SinceLast();

  uint64_t result = 0;
  result = thrust::reduce(
      thrust::device_ptr<int>(dev_results),
      thrust::device_ptr<int>(dev_results + m));

  Log() << "Reduce done " << timer->SinceLast();

  CUCHECK(hipFree(dev_edges));
  CUCHECK(hipFree(dev_pointers));
  CUCHECK(hipFree(dev_results));
  
  Log() << "Free done " << timer->SinceLast();

  delete timer;

  return result;
}
